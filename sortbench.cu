#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void prefix_scan_8_0_shmem(unsigned char *keys, int nitems, int *pfxs) {
    __shared__ int sh_pfxs[256];

    if (threadIdx.y < 8)
        sh_pfxs[threadIdx.y * 32 + threadIdx.x] = 0;

    __syncthreads();

    int blksz = blockDim.x * blockDim.y;
    int cap = nitems * (blockIdx.x + 1);

    for (int i = threadIdx.y * 32 + threadIdx.x + nitems * blockIdx.x;
         i < cap; i += blksz) {
        int value = keys[i];
        atomicAdd(sh_pfxs + value, 1);
    }

    __syncthreads();

    if (threadIdx.y < 8) {
        int off = threadIdx.y * 32 + threadIdx.x;
        atomicAdd(pfxs + off, sh_pfxs[off]);
    }
}

#define GRP_RDX_FACTOR (GRPSZ / RDXSZ)
#define GRP_BLK_FACTOR (GRPSZ / BLKSZ)
#define GRPSZ 8192
#define RDXSZ 256
#define BLKSZ 512

__global__
void prefix_scan(unsigned short *keys, int *pfxs, const int shift) {
    const int tid = threadIdx.y * 32 + threadIdx.x;
    __shared__ int shr_pfxs[BLKSZ];

    shr_pfxs[tid] = 0;
    __syncthreads();
    int i = tid + GRPSZ * blockIdx.x;

    for (int j = 0; j < GRP_BLK_FACTOR; j++) {
        int value = (keys[i] >> shift) && 0xff;
        atomicAdd(shr_pfxs + value, 1);
        i += BLKSZ;
    }

    __syncthreads();
    pfxs[tid + BLKSZ * blockIdx.x] = shr_pfxs[tid];
}

__global__
void prefix_scan_8_0_shmem_shortseg(unsigned char *keys, int *pfxs) {
    const int tid = threadIdx.y * 32 + threadIdx.x;
    __shared__ int shr_pfxs[RDXSZ];

    if (tid < RDXSZ) shr_pfxs[tid] = 0;
    __syncthreads();

    // TODO: this introduces a hard upper limit of 512M keys (3GB) sorted in a
    // pass. It'll be a while before we get the 8GB cards needed to do this.
    int i = tid + GRPSZ * blockIdx.x;

    for (int j = 0; j < GRP_BLK_FACTOR; j++) {
        int value = keys[i];
        atomicAdd(shr_pfxs + value, 1);
        i += BLKSZ;
    }

    __syncthreads();
    if (tid < RDXSZ) pfxs[tid + RDXSZ * blockIdx.x] = shr_pfxs[tid];
}

__global__
void crappy_split(int *pfxs, int *pfxs_out) {
    const int blksz = 256;
    const int tid = threadIdx.y * 32 + threadIdx.x;
    int i = blksz * (tid + blockIdx.x * blksz);
    int i_bound = i + blksz;
    int val = 0;
    for (; i < i_bound; i++) {
        pfxs_out[i] = val;
        val += pfxs[i];
    }
}

__global__
void better_split(int *pfxs_out, const int *pfxs) {
    // This one must be launched as 32x1, regardless of BLKSZ.
    const int tid = threadIdx.x;
    const int tid5 = tid << 5;
    __shared__ int swap[1024];

    int base = RDXSZ * 32 * blockIdx.x;

    int value = 0;

    // Performs a fast "split" (don't know why I called it that, will rename
    // soon). For each entry in pfxs (corresponding to the number of elements
    // per radix in a group), this writes the exclusive prefix sum for that
    // group. This is in fact a bunch of serial prefix sums in parallel, and
    // not a parallel prefix sum.
    //
    // The contents of 32 group radix counts are loaded in 32-element chunks
    // into shared memory, rotated by 1 unit each group to avoid bank
    // conflicts. Each thread in the warp sums across each group serially,
    // updating the values as it goes, then the results are written coherently
    // to global memory.
    //
    // This leaves the processor extremely compute-starved, as this only allows
    // 12 warps per SM. It might be better to halve the chunk size and lose
    // some coalescing efficiency; need to benchmark. It's a relatively cheap
    // step overall though.

    for (int j = 0; j < 8; j++) {
        int jj = j << 5;
        for (int i = 0; i < 32; i++) {
            int base_offset = (i << 8) + jj + base + tid;
            int swap_offset = (i << 5) + ((i + tid) & 0x1f);
            swap[swap_offset] = pfxs[base_offset];
        }

#pragma unroll
        for (int i = 0; i < 32; i++) {
            int swap_offset = tid5 + ((i + tid) & 0x1f);
            int tmp = swap[swap_offset];
            swap[swap_offset] = value;
            value += tmp;
        }

        for (int i = 0; i < 32; i++) {
            int base_offset = (i << 8) + jj + base + tid;
            int swap_offset = (i << 5) + ((i + tid) & 0x1f);
            pfxs_out[base_offset] = swap[swap_offset];
        }
    }
}

__global__
void prefix_sum(int *pfxs, int nitems, int *out_pfxs, int *out_sums) {
    // Needs optimizing (later). Should be rolled into split.
    // Must launch 32x8.
    const int tid = threadIdx.y * 32 + threadIdx.x;
    const int blksz = 256;
    int val = 0;
    for (int i = tid; i < nitems; i += blksz) val += pfxs[i];

    out_pfxs[tid] = val;

    // I know there's a better way to implement this summing network,
    // but it's not a time-critical piece of code.
    __shared__ int sh_pfxs[blksz];
    sh_pfxs[tid] = val;
    val = 0;
    __syncthreads();
    // Intentionally exclusive indexing here, val{0} should be 0
    for (int i = 0; i < tid; i++) val += sh_pfxs[i];
    out_sums[tid] = val;

    // Here we shift things over by 1, to make retrieving the
    // indices and differences easier in the sorting step.
    int i;
    for (i = tid; i < nitems; i += blksz) {
        int t = pfxs[i];
        pfxs[i] = val;
        val += t;
    }
    // Now write the last column and we're done.
    pfxs[i] = val;
}

__global__
void sort_8(unsigned char *keys, int *sorted_keys, int *pfxs) {
    const int tid = threadIdx.y * 32 + threadIdx.x;
    const int blk_offset = GRPSZ * blockIdx.x;
    __shared__ int shr_pfxs[RDXSZ];

    if (tid < RDXSZ) shr_pfxs[tid] = pfxs[RDXSZ * blockIdx.x + tid];
    __syncthreads();

    int i = tid;
    for (int j = 0; j < GRP_BLK_FACTOR; j++) {
        int value = keys[i+blk_offset];
        int offset = atomicAdd(shr_pfxs + value, 1);
        sorted_keys[offset] = value;
        i += BLKSZ;
    }
}

#undef BLKSZ
#define BLKSZ 1024
__global__
void sort_8_a(unsigned char *keys, int *sorted_keys,
              const int *pfxs, const int *split) {
    const int tid = threadIdx.y * 32 + threadIdx.x;
    const int blk_offset = GRPSZ * blockIdx.x;
    __shared__ int shr_offs[RDXSZ];
    __shared__ int defer[GRPSZ];

    const int pfx_i = RDXSZ * blockIdx.x + tid;
    if (tid < RDXSZ) shr_offs[tid] = split[pfx_i];
    __syncthreads();

    for (int i = tid; i < GRPSZ; i += BLKSZ) {
        int value = keys[i+blk_offset];
        int offset = atomicAdd(shr_offs + value, 1);
        defer[offset] = value;
    }
    __syncthreads();

    // This calculation is a bit odd.
    //
    // For a given radix value 'r', shr_offs[r] currently holds the first index
    // of the *next* radix in defer[] (i.e.  if there are 28 '0'-radix values
    // in defer[], shr_offs[0]==28). We want to get back to a normal exclusive
    // prefix, so we subtract shr_offs[0] from everything.
    //
    // In the next block, we want to be able to find the correct position for a
    // value in defer[], given that value's index 'i' and its radix 'r'. This
    // requires two values: the destination index in sorted_keys[] of the first
    // value in the group with radix 'r' (given by pfxs[BASE + r]), and the
    // number of radix-'r' values before this one in defer[]. So, ultimately,
    // we want an equation in the inner loop below that looks like this:
    //
    //      int dst_offset = pfxs[r] + i - (shr_offs[r] - shr_offs[0]);
    //      sorted_keys[dst_offset] = defer[i];
    //
    // Of course, this generates tons of memory lookups and bank conflicts so
    // we precombine some of this here.
    int off0 = shr_offs[0];
    if (tid < RDXSZ) shr_offs[tid] = pfxs[0] - (shr_offs[tid] - off0);
    __syncthreads();

    int i = tid;
#pragma unroll
    for (int j = 0; j < GRP_BLK_FACTOR; j++) {
        int value = defer[i];
        int offset = shr_offs[value] + i;
        sorted_keys[offset] = value;
        i += BLKSZ;
    }
}



__global__
void prefix_scan_8_0_shmem_lessconf(unsigned char *keys, int nitems, int *pfxs) {
    __shared__ int sh_pfxs_banked[256][32];

    for (int i = threadIdx.y; i < 256; i += blockDim.y)
        sh_pfxs_banked[i][threadIdx.x] = 0;

    __syncthreads();

    int blksz = blockDim.x * blockDim.y;
    int cap = nitems * (blockIdx.x + 1);

    for (int i = threadIdx.y * 32 + threadIdx.x + nitems * blockIdx.x;
         i < cap; i += blksz) {
        int value = keys[i];
        atomicAdd(&(sh_pfxs_banked[value][threadIdx.x]), 1);
    }

    __syncthreads();

    for (int i = threadIdx.y; i < 256; i += blockDim.y) {
        for (int j = 16; j > 0; j = j >> 1)
            if (j > threadIdx.x)
                sh_pfxs_banked[i][threadIdx.x] += sh_pfxs_banked[i][j+threadIdx.x];
        __syncthreads();
    }

    if (threadIdx.y < 8) {
        int off = threadIdx.y * 32 + threadIdx.x;
        atomicAdd(pfxs + off, sh_pfxs_banked[off][0]);
    }

}

__global__
void prefix_scan_5_0_popc(unsigned char *keys, int nitems, int *pfxs) {
    __shared__ int sh_pfxs[32];

    if (threadIdx.y == 0) sh_pfxs[threadIdx.x] = 0;

    __syncthreads();

    int blksz = blockDim.x * blockDim.y;
    int cap = nitems * (blockIdx.x + 1);

    int sum = 0;

    for (int i = threadIdx.y * 32 + threadIdx.x + nitems * blockIdx.x;
         i < cap; i += blksz) {

        int value = keys[i];
        int test = __ballot(value & 1);
        if (!(threadIdx.x & 1)) test = ~test;

        int popc_res = __ballot(value & 2);
        if (!(threadIdx.x & 2)) popc_res = ~popc_res;
        test &= popc_res;

        popc_res = __ballot(value & 4);
        if (!(threadIdx.x & 4)) popc_res = ~popc_res;
        test &= popc_res;

        popc_res = __ballot(value & 8);
        if (!(threadIdx.x & 8)) popc_res = ~popc_res;
        test &= popc_res;

        popc_res = __ballot(value & 16);
        if (!(threadIdx.x & 16)) popc_res = ~popc_res;
        test &= popc_res;

        sum += __popc(test);
    }

    atomicAdd(sh_pfxs + threadIdx.x + 0,   sum);
    __syncthreads();

    if (threadIdx.y == 0) {
        int off = threadIdx.x;
        atomicAdd(pfxs + off, sh_pfxs[off]);
    }
}


__global__
void prefix_scan_8_0_popc(unsigned char *keys, int nitems, int *pfxs) {
    __shared__ int sh_pfxs[256];

    if (threadIdx.y < 8)
        sh_pfxs[threadIdx.y * 32 + threadIdx.x] = 0;

    __syncthreads();

    int blksz = blockDim.x * blockDim.y;
    int cap = nitems * (blockIdx.x + 1);

    int sum_000 = 0;
    int sum_001 = 0;
    int sum_010 = 0;
    int sum_011 = 0;
    int sum_100 = 0;
    int sum_101 = 0;
    int sum_110 = 0;
    int sum_111 = 0;

    for (int i = threadIdx.y * 32 + threadIdx.x + nitems * blockIdx.x;
         i < cap; i += blksz) {

        int value = keys[i];
        int test_000 = __ballot(value & 1);
        if (!(threadIdx.x & 1)) test_000 = ~test_000;

        int popc_res = __ballot(value & 2);
        if (!(threadIdx.x & 2)) popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 4);
        if (!(threadIdx.x & 4)) popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 8);
        if (!(threadIdx.x & 8)) popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 16);
        if (!(threadIdx.x & 16)) popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 32);
        int test_001 = test_000 & popc_res;
        popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 64);
        int test_010 = test_000 & popc_res;
        int test_011 = test_001 & popc_res;
        popc_res = ~popc_res;
        test_000 &= popc_res;
        test_001 &= popc_res;

        popc_res = __ballot(value & 128);
        int test_100 = test_000 & popc_res;
        int test_101 = test_001 & popc_res;
        int test_110 = test_010 & popc_res;
        int test_111 = test_011 & popc_res;
        popc_res = ~popc_res;
        test_000 &= popc_res;
        test_001 &= popc_res;
        test_010 &= popc_res;
        test_011 &= popc_res;

        sum_000 += __popc(test_000);
        sum_001 += __popc(test_001);
        sum_010 += __popc(test_010);
        sum_011 += __popc(test_011);
        sum_100 += __popc(test_100);
        sum_101 += __popc(test_101);
        sum_110 += __popc(test_110);
        sum_111 += __popc(test_111);
    }

    atomicAdd(sh_pfxs + (threadIdx.x + 0),   sum_000);
    atomicAdd(sh_pfxs + (threadIdx.x + 32),  sum_001);
    atomicAdd(sh_pfxs + (threadIdx.x + 64),  sum_010);
    atomicAdd(sh_pfxs + (threadIdx.x + 96),  sum_011);
    atomicAdd(sh_pfxs + (threadIdx.x + 128), sum_100);
    atomicAdd(sh_pfxs + (threadIdx.x + 160), sum_101);
    atomicAdd(sh_pfxs + (threadIdx.x + 192), sum_110);
    atomicAdd(sh_pfxs + (threadIdx.x + 224), sum_111);

    __syncthreads();

    if (threadIdx.y < 8) {
        int off = threadIdx.y * 32 + threadIdx.x;
        atomicAdd(pfxs + off, sh_pfxs[off]);
    }
}

